#include "hip/hip_runtime.h"
﻿#include"gsBlur.h"

using namespace std;
using namespace cv;

__global__ void kernel_fun_gs(const Mat& img1,Mat& img2,Mat& img3) {
	//img1 from cpu,img3; img2 from gpu
	//1. cuda malloc img2,img3
	//2. copy img1 to img2
	//3. img2 <--> thread(x,y) inflection
	//4. cope img2 with gs kernel(make gs kernel a contant)
	//5. img3=copty(img2，from gpu to cpu)

	return;
}



int MAXV = 150;
int Light[] = { 10,20 };	//Light若为const，被调时会报错
Mat src,res,mtmp;

void onChange(int, void*) {//trackbar回调函数
	// 填充算子
	//mtmp = Mat::ones(img.size(), img.type());
	//相加的话会偏亮，颜色也由三色合成的强分量方向改变。注意这里必须用Light，也就是监听函数的参数
	mtmp = Scalar(Light[1], Light[0], Light[0]);	
	add(src, mtmp, res);	//加法，亮度增强
	imshow("show", res);//输出语句要放在回调函数中，否则拖拉bar时不起作用(因为回调函数会被反复执行，而trackbar函数只被调用一次)
}

void trackbar_test(Mat &img) {
	namedWindow("show", WINDOW_AUTOSIZE);
	mtmp = Mat::zeros(img.size(), img.type());	//这里必须给全局变量分配空间，否则出问题
	res = Mat::zeros(img.size(), img.type());
	src=img.clone();
	//建立TrackBar监听
	createTrackbar("bar", "show", Light, MAXV, onChange);
	//onChange(10, 0);	//初始时的bar值(light=10)，可加可不加
}


void onChange_pro(int val, void* img) {//trackbar回调函数
	Mat src = *((Mat*)img);
	Mat tmp = Mat::zeros(src.size(), src.type());
	Mat res = Mat::zeros(src.size(), src.type());
	tmp = Scalar(val,val,val);
	add(src, tmp,res);
	//namedWindow("res", WINDOW_AUTOSIZE);	//该处可以省略,window放在点击事件中
	imshow("res", res);
}

void trackbar_test_pro(Mat& img) {	//利用好creatTrackBar的参数传递
	int val=10; 
	namedWindow("res", WINDOW_AUTOSIZE);	//这个不能省略，且"res"要和下面的窗口名一致
	createTrackbar("bar", "res", &val, 150, onChange_pro, (void*)&img);
	//onChange_pro(10, &img);
}



int mainxx() {
	//string path = "..\\source\\picSc.jpg";	//注意路径的写法(相对路径时，加上当前入口路径)
	//Mat img = imread(path);	//可以设置读入方式，灰度图，HSV颜色空间，含透明通道等
	//if (img.empty()) {		//如果路径不对或者为全黑图
	//	cerr << "Maybe the img path is false." << endl;
	//	return -1;
	//}
	//D:\VS2019\workplace\cudaTest\source
	//namedWindow("img windows", WINDOW_FREERATIO);  //根据图片大小自动调整窗口大小
	//imshow("img windows", pic1);	//将pic1图片显示到img windows中
	//imshow("img", img);
	//waitKey(3000);	//显示3000ms,即3s
	//destroyAllWindows();	//关闭所有显示窗口


	// trackBarTest
	/*string path = "..\\source\\picSc.jpg";
	Mat img = imread(path);
	trackbar_test(img);
	waitKey(0);
	destroyAllWindows(); */


	//TrackBar优化
	/*string path = "..\\source\\picSc.jpg";
	Mat img = imread(path);
	trackbar_test_pro(img);*/

	//测试颜色
	/*Mat tmp = Mat::zeros(Size(100, 100),CV_8UC3);
	tmp = Scalar(40, 40, 200);
	imshow("tmp", tmp);*/



	waitKey(0);
	destroyAllWindows();

	return 0;
}





